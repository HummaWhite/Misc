#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <cstdlib>
#include <iomanip>
#include <cmath>
#include <ctime>

#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

#define asmv asm __volatile

struct Body {
	float3 pos;
	float3 vel;
};

void CHECK_ERROR(const std::string &msg = "") {
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "[%s] Launch failed: %s\n", msg.c_str(), hipGetErrorString(cudaStatus));
	}
}

std::string cudaPropToStr(const hipDeviceProp_t &prop) {
	std::stringstream ss;

	ss << "[Name = " << prop.name << ", ";
	ss << "GlobalMemSize = " << prop.totalGlobalMem / 1048676 << "MB, ";
	ss << "SharedMemPerBlock = " << prop.sharedMemPerBlock / 1024 << "KB, ";
	ss << "ConstMemSize = " << prop.totalConstMem / 1024 << "KB, ";
	ss << "RegsPerBlock = " << prop.regsPerBlock << ", ";
	ss << "WarpSize = " << prop.warpSize << ", ";
	ss << "MemPitch = " << prop.memPitch << ", ";
	ss << "MaxThreadsPerBlock = " << prop.maxThreadsPerBlock << ", ";
	ss << "MaxThreadsDim = {" << prop.maxThreadsDim[0] << ", ";
	ss << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << "}, ";
	ss << "MaxGridSize = {" << prop.maxGridSize[0] << ", ";
	ss << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << "}]";

	return ss.str();
}

void QUERY_DEVICE_PROP() {
	int nGPU;
	hipGetDeviceCount(&nGPU);
	std::cout << "[Device Count] " << nGPU << "\n";

	for (int i = 0; i < nGPU; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		std::cout << "[Device " << i << " = " << cudaPropToStr(prop) << "]\n";
	}
}

/*
 * Do not modify this function. A constraint of this exercise is
 * that it remain a host function.
 */

float uniformFloat() {
	return ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
}

float3 uniformFloat3() {
	return { uniformFloat(), uniformFloat(), uniformFloat() };
}

void randomizeBodies(float3 *pos, float3 *vel, int n) {
	for (int i = 0; i < n; i++) {
		pos[i] = uniformFloat3();
		vel[i] = uniformFloat3();
	}
}

/*
 * This function calculates the gravitational impact of all bodies in the system
 * on all others, but does not update their positions.
 */

std::string float3ToString(const float3 &v) {
	std::stringstream ss;
	ss << std::setprecision(12) << "[" << v.x << ", " << v.y << ", " << v.z << "]";
	return ss.str();
}

__device__ float3 operator + (const float3 &a, const float3 &b) {
	return { a.x + b.x, a.y + b.y, a.z + b.z };
}

__device__ float3 operator - (const float3 &a, const float3 &b) {
	return { a.x - b.x, a.y - b.y, a.z - b.z };
}

__device__ float3 operator * (const float3 &a, float v) {
	return { a.x * v, a.y * v, a.z * v };
}

__device__ void operator += (float3 &a, const float3 &b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}

__forceinline__ __device__ float dot(const float3 &a, const float3 &b) {
	const float SOFTENING = 1e-9f;
	float res = SOFTENING;
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.x), "f"(b.x), "f"(res));
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.y), "f"(b.y), "f"(res));
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.z), "f"(b.z), "f"(res));
	return res;
}

__forceinline__ __device__ void bodyGetAccelPairwise(float3 posI, float3 posJ, float3 &acc) {
	// 3fsub + 1frsqrt + 2fmul + 6fma = 12 float instructions
	float3 dp = posJ - posI;

	// 4fadd, 3fmul with regular calculation method
	// By using fma, only 3 instructions
	float distSqr = dot(dp, dp);
	float invDist = rsqrtf(distSqr);
	float invDist3 = invDist * invDist * invDist;

	acc += dp * invDist3;
}

__forceinline__ __device__ void bodyGatherAccelTiled(float3 pos, float3 &acc) {
	extern __shared__ float3 posShared[];
#pragma unroll 32
	for (int i = 0; i < blockDim.x; i += 2) {
		bodyGetAccelPairwise(pos, posShared[i + 1], acc);
		bodyGetAccelPairwise(pos, posShared[i], acc);
	}
}

__global__ void bodyUpdateVelocity(float3 *pos, float3 *vel, float dt, int n) {
	int bodyId = blockIdx.x * blockDim.x + threadIdx.x;
	if (bodyId >= n) return;
	extern __shared__ float3 posShared[];

	float3 acc = { 0.0f, 0.0f, 0.0f };
	float3 p = pos[bodyId];

	for (int i = 0; i < gridDim.x; i++) {
		// For all threads in the block, async load tiled position
		// data to shared memory
		int id = i * blockDim.x + threadIdx.x;
		__syncthreads();
		posShared[threadIdx.x] = pos[id];
		__syncthreads();
		bodyGatherAccelTiled(p, acc);
	}
	vel[bodyId] += acc * dt;
}

__global__ void bodyUpdatePosition(float3 *pos, float3 *vel, float dt, int n) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n) return;

	pos[i] += vel[i] * dt;
}

int main(const int argc, const char *argv[]) {
	//QUERY_DEVICE_PROP();

	int nBodies = 2 << 11;
	int salt = 0;
	if (argc > 1) nBodies = 2 << std::atoi(argv[1]);

	if (argc > 2) salt = std::atoi(argv[2]);

	const float dt = 0.01f; // time step
	const int nIters = 10;  // simulation iterations

	float3 *posHost = new float3[nBodies];
	float3 *velHost = new float3[nBodies];
	float *buf = new float[nBodies * 6];

	randomizeBodies(posHost, velHost, nBodies); // Init pos / vel data

	size_t dataSize = nBodies * sizeof(float3);

	hipSetDevice(0);

	float3 *posDevice, *velDevice;
	hipMalloc<float3>(&posDevice, dataSize);
	hipMalloc<float3>(&velDevice, dataSize);
	hipMemcpy(posDevice, posHost, dataSize, hipMemcpyKind::hipMemcpyHostToDevice);
	hipMemcpy(velDevice, velHost, dataSize, hipMemcpyKind::hipMemcpyHostToDevice);

	const int blockSize = 128;
	const int blockNum = int(std::ceil(double(nBodies) / blockSize));
	const int sharedMemSize = blockSize * sizeof(float3);

	double totalTime = 0.0;

	/*******************************************************************/
	// Do not modify these 2 lines of code.
	for (int iter = 0; iter < nIters; iter++) {
		StartTimer();
		/*******************************************************************/

		bodyUpdateVelocity<<<blockNum, blockSize, sharedMemSize>>>(posDevice, velDevice, dt, nBodies);
		bodyUpdatePosition<<<blockNum, blockSize>>>(posDevice, velDevice, dt, nBodies);

		// Well, this is the solution, though I still want to put
		// it out of the loop
		if (iter == nIters - 1) {
			hipDeviceSynchronize();
			// I don't think this should be taken into account,
			// it's just moving results from GPU to host memory
			// and data reorganize
			totalTime += GetTimer() / 1000.0;
			hipMemcpy(posHost, posDevice, dataSize, hipMemcpyKind::hipMemcpyDeviceToHost);
			hipMemcpy(velHost, velDevice, dataSize, hipMemcpyKind::hipMemcpyDeviceToHost);

			for (int j = 0; j < nBodies; j++) {
				*(float3*)(buf + j * 6) = posHost[j];
				*(float3*)(buf + j * 6 + 3) = velHost[j];
			}
			StartTimer();
		}

		/*******************************************************************/
		// Do not modify the code in this section.
		const double tElapsed = GetTimer() / 1000.0;
		totalTime += tElapsed;
	}

	double avgTime = totalTime / (double)(nIters);
	float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
	checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
	checkAccuracy(buf, nBodies);
	printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
	salt += 1;
#endif
	/*******************************************************************/

	std::cout << float3ToString(posHost[9]) << "\n";

	hipFree(posDevice);
	hipFree(velDevice);

	delete[] posHost;
	delete[] velHost;
	delete[] buf;
}
