#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <cstdlib>
#include <iomanip>
#include <cmath>
#include <ctime>

#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

#define asmv asm __volatile

void CHECK_ERROR(const std::string &msg = "") {
	hipError_t cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "[%s] Launch failed: %s\n", msg.c_str(), hipGetErrorString(cudaStatus));
	}
}

float uniformFloat() {
	return ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
}

float3 uniformFloat3() {
	return { uniformFloat(), uniformFloat(), uniformFloat() };
}

void randomizeBodies(float3 *pos, float3 *vel, int n) {
	for (int i = 0; i < n; i++) {
		pos[i] = uniformFloat3();
		vel[i] = uniformFloat3();
	}
}

std::string float3ToString(const float3 &v) {
	std::stringstream ss;
	ss << std::setprecision(12) << "[" << v.x << ", " << v.y << ", " << v.z << "]";
	return ss.str();
}

__device__ float3 operator + (const float3 &a, const float3 &b) {
	return { a.x + b.x, a.y + b.y, a.z + b.z };
}

__device__ float3 operator - (const float3 &a, const float3 &b) {
	return { a.x - b.x, a.y - b.y, a.z - b.z };
}

__device__ float3 operator * (const float3 &a, float v) {
	return { a.x * v, a.y * v, a.z * v };
}

__device__ void operator += (float3 &a, const float3 &b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}

__forceinline__ __device__ float dot(const float3 &a, const float3 &b) {
	const float SOFTENING = 1e-9f;
	float res = SOFTENING;
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.x), "f"(b.x), "f"(res));
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.y), "f"(b.y), "f"(res));
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.z), "f"(b.z), "f"(res));
	return res;
}

__forceinline__ __device__ void bodyGetAccelPairwise(float3 posI, float3 posJ, float3 &acc) {
	// 3fsub + 1frsqrt + 2fmul + 6fma = 12 float instructions
	float3 dp = posJ - posI;

	// 4fadd, 3fmul with regular calculation method
	// By using fma, only 3 instructions
	float distSqr = dot(dp, dp);
	float invDist = rsqrtf(distSqr);
	float invDist3 = invDist * invDist * invDist;

	acc += dp * invDist3;
}

__forceinline__ __device__ void bodyGatherAccelTiled(float3 pos, float3 &acc) {
	extern __shared__ float3 posShared[];
#pragma unroll 32
	for (int i = 0; i < blockDim.x; i += 2) {
		bodyGetAccelPairwise(pos, posShared[i + 1], acc);
		bodyGetAccelPairwise(pos, posShared[i], acc);
	}
}

__global__ void bodyCalcDeltaVelocity(float3 *pos, float3 *dvel, float dt, int n, int offset) {
	int bodyId = blockIdx.x * blockDim.x + threadIdx.x;
	if (bodyId >= n) return;
	extern __shared__ float3 posShared[];

	float3 acc = { 0.0f, 0.0f, 0.0f };
	float3 p = pos[bodyId];

	for (int i = 0; i < gridDim.x / 4; i++) {
		// For all threads in the block, async load tiled position
		// data to shared memory
		int id = i * blockDim.x + threadIdx.x + offset;
		__syncthreads();
		posShared[threadIdx.x] = pos[id];
		__syncthreads();
		bodyGatherAccelTiled(p, acc);
	}
	dvel[bodyId] = acc * dt;
}

__global__ void bodyUpdateVelAndPos(float3 *pos, float3 *vel, float3 *deltaVel, float dt, int n) {
	int bodyId = blockIdx.x * blockDim.x + threadIdx.x;
	if (bodyId >= n) return;
#pragma unroll
	for (int i = 0; i < 4; i++) {
		vel[bodyId] += deltaVel[i * n + bodyId];
	}
	pos[bodyId] += vel[bodyId] * dt;
}

int main(const int argc, const char *argv[]) {
	//QUERY_DEVICE_PROP();

	int nBodies = 2 << 11;
	int salt = 0;
	if (argc > 1) nBodies = 2 << std::atoi(argv[1]);

	if (argc > 2) salt = std::atoi(argv[2]);

	const float dt = 0.01f; // time step
	const int nIters = 10;  // simulation iterations

	float3 *posHost = new float3[nBodies];
	float3 *velHost = new float3[nBodies];
	float *buf = new float[nBodies * 6];

	randomizeBodies(posHost, velHost, nBodies); // Init pos / vel data

	size_t dataSize = nBodies * sizeof(float3);

	for (int i = 0; i < 4; i++) {
		hipSetDevice(i);
		hipDeviceEnablePeerAccess(i ^ 1, 0);
	}

	float3 *posDevice[4], *velDevice[4];
	float3 *deltaVel[4];
	for (int i = 0; i < 4; i++) {
		hipSetDevice(i);
		hipMalloc<float3>(&posDevice[i], dataSize);
		hipMalloc<float3>(&velDevice[i], dataSize);
		hipMalloc<float3>(&deltaVel[i], dataSize * 4);
		hipMemcpy(posDevice[i], posHost, dataSize, hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(velDevice[i], velHost, dataSize, hipMemcpyKind::hipMemcpyHostToDevice);
	}

	const int blockSize = 128;
	const int blockNum = int(std::ceil(double(nBodies) / blockSize));
	const int sharedMemSize = blockSize * sizeof(float3);

	double totalTime = 0.0;

	/*******************************************************************/
	// Do not modify these 2 lines of code.
	for (int iter = 0; iter < nIters; iter++) {
		StartTimer();
		/*******************************************************************/

		for (int i = 0; i < 4; i++) {
			hipSetDevice(i);
			bodyCalcDeltaVelocity<<<blockNum, blockSize, sharedMemSize>>>(posDevice[i], deltaVel[i] + nBodies * i, dt, nBodies, nBodies / 4 * i);
		}
		hipDeviceSynchronize();

		for (int i = 0; i < 4; i++) {
			hipSetDevice(i);
			int d = i ^ 1;
			hipMemcpyPeerAsync(deltaVel[d] + nBodies * i, d, deltaVel[i] + nBodies * i, i, dataSize);
		}

		for (int i = 0; i < 4; i++) {
			hipSetDevice(i);
			int d = i ^ 2;
			hipMemcpy(deltaVel[d] + nBodies * (i & 2), deltaVel[i] + nBodies * (i & 2), dataSize * 2, hipMemcpyKind::hipMemcpyDeviceToDevice);
		}

		for (int i = 0; i < 4; i++) {
			hipSetDevice(i);
			bodyUpdateVelAndPos<<<blockNum, blockSize>>>(posDevice[i], velDevice[i], deltaVel[i], dt, nBodies);
		}

		// Well, this is the solution, though I still want to put
		// it out of the loop
		if (iter == nIters - 1) {
			// I don't think this should be taken into account,
			// it's just moving results from GPU to host memory
			// and data reorganize
			hipDeviceSynchronize();
			totalTime += GetTimer() / 1000.0;
			hipSetDevice(0);
			hipMemcpy(posHost, posDevice[0], dataSize, hipMemcpyKind::hipMemcpyDeviceToHost);
			hipMemcpy(velHost, velDevice[0], dataSize, hipMemcpyKind::hipMemcpyDeviceToHost);

			for (int j = 0; j < nBodies; j++) {
				*(float3*)(buf + j * 6) = posHost[j];
				*(float3*)(buf + j * 6 + 3) = velHost[j];
			}
			StartTimer();
		}

		/*******************************************************************/
		// Do not modify the code in this section.
		const double tElapsed = GetTimer() / 1000.0;
		totalTime += tElapsed;
	}

	double avgTime = totalTime / (double)(nIters);
	float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
	checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
	checkAccuracy(buf, nBodies);
	printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
	salt += 1;
#endif
	/*******************************************************************/

	std::cout << float3ToString(posHost[9]) << "\n";

	for (int i = 0; i < 4; i++) {
		hipSetDevice(i);
		hipFree(posDevice[i]);
		hipFree(velDevice[i]);
		hipFree(deltaVel[i]);
	}

	delete[] posHost;
	delete[] velHost;
	delete[] buf;
}
