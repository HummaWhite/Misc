#include "hip/hip_runtime.h"
#include <iostream>
#include <sstream>
#include <cstdlib>
#include <iomanip>
#include <cmath>
#include <ctime>

#include "timer.h"
#include "check.h"
#include <hip/hip_runtime.h>

float uniformFloat() {
	// 产生随机数
	return ((float)rand() / RAND_MAX) * 2.0f - 1.0f;
}

float3 uniformFloat3() {
	return { uniformFloat(), uniformFloat(), uniformFloat() };
}

void randomizeBodies(float3 *pos, float3 *vel, int n) {
	// 随机初始化body数据
	for (int i = 0; i < n; i++) {
		pos[i] = uniformFloat3();
		vel[i] = uniformFloat3();
	}
}
// 重载运算符以支持向量运算
__device__ float3 operator + (const float3 &a, const float3 &b) {
	return { a.x + b.x, a.y + b.y, a.z + b.z };
}

__device__ float3 operator - (const float3 &a, const float3 &b) {
	return { a.x - b.x, a.y - b.y, a.z - b.z };
}

__device__ float3 operator * (const float3 &a, float v) {
	return { a.x * v, a.y * v, a.z * v };
}

__device__ void operator += (float3 &a, const float3 &b) {
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
}

__forceinline__ __device__ float dot(const float3 &a, const float3 &b) {
	// 使用fma指令优化向量内积
	const float SOFTENING = 1e-9f;
	float res = SOFTENING;
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.x), "f"(b.x), "f"(res));
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.y), "f"(b.y), "f"(res));
	asm("fma.rn.f32         %0, %1, %2, %3;" : "=f"(res) : "f"(a.z), "f"(b.z), "f"(res));
	return res;
}

__forceinline__ __device__ void bodyGetAccelPairwise(float3 posI, float3 posJ, float3 &acc) {
	// 计算一对body之间的加速度
	// 3fsub + 1frsqrt + 2fmul + 6fma = 12 float instructions
	float3 dp = posJ - posI;

	// 4fadd, 3fmul with regular calculation method
	// By using fma, only 3 instructions
	float distSqr = dot(dp, dp);
	float invDist = rsqrtf(distSqr);
	float invDist3 = invDist * invDist * invDist;

	acc += dp * invDist3;
}

__forceinline__ __device__ void bodyGatherAccelTiled(float3 pos, float3 &acc) {
	// 使用共享内存计算一批body对某个body的加速度总和
	extern __shared__ float3 posShared[];
	// 使用预处理命令展开循环
#pragma unroll 32
	for (int i = 0; i < blockDim.x; i += 2) {
		bodyGetAccelPairwise(pos, posShared[i + 1], acc);
		bodyGetAccelPairwise(pos, posShared[i], acc);
	}
}

__global__ void bodyCalcDeltaVelocity(float3 *pos, float3 *dvel, float dt, int n, int offset) {
	// 计算
	int bodyId = blockIdx.x * blockDim.x + threadIdx.x;
	if (bodyId >= n) return;
	extern __shared__ float3 posShared[];

	float3 acc = { 0.0f, 0.0f, 0.0f };
	float3 p = pos[bodyId];

	for (int i = 0; i < gridDim.x / 2; i++) {
		// 加载blockDim.x个body位置进入共享内存
		int id = i * blockDim.x + threadIdx.x + offset;
		__syncthreads();
		posShared[threadIdx.x] = pos[id];
		__syncthreads();
		// 需要等待全部线程将数据载入共享内存后才能进行计算，反之亦然
		bodyGatherAccelTiled(p, acc);
	}
	// 计算速度增量
	dvel[bodyId] = acc * dt;
}

__global__ void bodyUpdateVelAndPos(float3 *pos, float3 *vel, float3 *dvSelf, float3 *dvOppo, float dt, int n) {
	// 更新body的速度和坐标
	int bodyId = blockIdx.x * blockDim.x + threadIdx.x;
	if (bodyId >= n) return;
	// 本GPU内计算的dv与另一个GPU计算的速度增量累加，更新速度
	vel[bodyId] += dvSelf[bodyId] + dvOppo[bodyId];
	// 更新坐标
	pos[bodyId] += vel[bodyId] * dt;
}

int main(const int argc, const char *argv[]) {
	int nBodies = 2 << 11;
	int salt = 0;
	if (argc > 1) nBodies = 2 << std::atoi(argv[1]);

	if (argc > 2) salt = std::atoi(argv[2]);

	const float dt = 0.01f; // time step
	const int nIters = 10;  // simulation iterations
	// 分配主存中的body坐标与速度的存储空间
	float3 *posHost = new float3[nBodies];
	float3 *velHost = new float3[nBodies];
	float *buf = new float[nBodies * 6];
	// 初始化body属性
	randomizeBodies(posHost, velHost, nBodies); // Init pos / vel data

	size_t dataSize = nBodies * sizeof(float3);
	const int devOffset = 0;

	for (int i = 0; i < 2; i++) {
		int acc;
		// 查询两个逻辑GPU是否满足peer access条件
		hipDeviceCanAccessPeer(&acc, devOffset + i, devOffset + (i ^ 1));
		if (acc) {
			// 若满足，互相设置为可访问对方内存地址空间
			hipSetDevice(devOffset + i);
			hipDeviceEnablePeerAccess(devOffset + (i ^ 1), 0);
		}
	}
	// 分配两个GPU上的存储空间
	float3 *posDevice[2], *velDevice[2];
	// deltaVel暂存速度变化量
	float3 *deltaVel[2];
	for (int i = 0; i < 2; i++) {
		hipSetDevice(i + devOffset);
		hipMalloc<float3>(&posDevice[i], dataSize);
		hipMalloc<float3>(&velDevice[i], dataSize);
		hipMalloc<float3>(&deltaVel[i], dataSize);
		// 复制body数据到GPU
		hipMemcpy(posDevice[i], posHost, dataSize, hipMemcpyKind::hipMemcpyHostToDevice);
		hipMemcpy(velDevice[i], velHost, dataSize, hipMemcpyKind::hipMemcpyHostToDevice);
	}
	// 确定线程块大小和线程块的数量
	const int blockSize = 128;
	const int blockNum = int(std::ceil(double(nBodies) / blockSize));
	const int sharedMemSize = blockSize * sizeof(float3);

	double totalTime = 0.0;

	/*******************************************************************/
	// Do not modify these 2 lines of code.
	for (int iter = 0; iter < nIters; iter++) {
		StartTimer();
		/*******************************************************************/
		// 计算两个GPU上分到的n/2个body对n个body形成的n个速度增量
		for (int i = 0; i < 2; i++) {
			hipSetDevice(i + devOffset);
			bodyCalcDeltaVelocity<<<blockNum, blockSize, sharedMemSize>>>(posDevice[i], deltaVel[i], dt, nBodies, i * nBodies / 2);
		}
		// 需要等待两个GPU都完成计算
		hipDeviceSynchronize();
		// 更新速度和坐标
		for (int i = 0; i < 2; i++) {
			hipSetDevice(i + devOffset);
			bodyUpdateVelAndPos<<<blockNum, blockSize>>>(posDevice[i], velDevice[i], deltaVel[i], deltaVel[i ^ 1], dt, nBodies);
		}

		// Well, this is the solution, though I still want to put
		// it out of the loop
		if (iter == nIters - 1) {
			// 最后一次迭代完成后同步CPU与GPU
			hipDeviceSynchronize();
			totalTime += GetTimer() / 1000.0;
			// 从GPU读回数据
			hipSetDevice(devOffset);
			hipMemcpy(posHost, posDevice[0], dataSize, hipMemcpyKind::hipMemcpyDeviceToHost);
			hipMemcpy(velHost, velDevice[0], dataSize, hipMemcpyKind::hipMemcpyDeviceToHost);
			// 调整数据存储方式
			for (int j = 0; j < nBodies; j++) {
				*(float3*)(buf + j * 6) = posHost[j];
				*(float3*)(buf + j * 6 + 3) = velHost[j];
			}
			StartTimer();
		}

		/*******************************************************************/
		// Do not modify the code in this section.
		const double tElapsed = GetTimer() / 1000.0;
		totalTime += tElapsed;
	}

	double avgTime = totalTime / (double)(nIters);
	float billionsOfOpsPerSecond = 1e-9 * nBodies * nBodies / avgTime;

#ifdef ASSESS
	checkPerformance(buf, billionsOfOpsPerSecond, salt);
#else
	checkAccuracy(buf, nBodies);
	printf("%d Bodies: average %0.3f Billion Interactions / second\n", nBodies, billionsOfOpsPerSecond);
	salt += 1;
#endif
	/*******************************************************************/
	// 释放所有已分配的内存
	for (int i = 0; i < 2; i++) {
		hipSetDevice(i + devOffset);
		hipFree(posDevice[i]);
		hipFree(velDevice[i]);
		hipFree(deltaVel[i]);
	}

	delete[] posHost;
	delete[] velHost;
	delete[] buf;
}
